#include "hip/hip_runtime.h"
//int B, int T, int C // 输入输出shape, 默认为 8, 1024, 768
//const float* inp // 输入x, shape为 [B, T, C]
//float* mean, float* rstd // 输入x的均值\miu, 标准差的倒数 1/\sigma
//const float* weight, const float* bias // 可学习的权重及偏置， 随机初始化后传入
//float* out // 输出, shape为 [B, T, C]

void layernorm_forward_cpu(float* out, float* mean, float* rstd, const float* inp, const float* weight, const float* bias, int B, int T, int C) {
    float eps = 1e-5f;
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            // seek to the input position inp[b, t, :]
            const float* x = inp + b * T * C + t * C;
            // calculate the mean
            float m = 0.0f;
            for (int i = 0; i < C; i ++) {
                m += x[i];
            }
            m = m/C;
            //calculate the variance (without any bias correction)
            float v = 0.0f;
            for (int i = 0; i < C; i++) {
                float xshift = x[i] - m;
                v += xshift * xshift;
            }
            v = v/C;
            // calculate the rstd
            float s = 1.0f / sqrtf(v + eps);
            // seek to the output position in out[b,t,:]
            float* out_bt = out + b * T * C + t * C;
            for (int i = 0; i < C; i++) {
                float n = (s * (x[i] - m)); // normalized output
                float o = n * weight[i] + bias[i]; // scale and shift it
                out_bt[i] = o; // write
            }
            // cache the mean and rstd for the backward pass later
            mean[b * T + t] = m;
            rstd[b * T + t] = s;
        }
    }
}

void layernorm_forward1(float* out, float* mean, float* rstd, const float* inp, const float* weight, const float* bias, int B, int T, int C, const int block_size){
    const int N = B * T;
    const int grid_size = ceil_div(N, block_size);
    layernorm_forward_kernel1<<<grid_size, block_size>>>(out, mean, rstd, inp, weight, bias, N, C);
    cudaCheck(hipGetLastError());
}

__global__ void mean_kernel(float* mean, const float* inp, int N, int C, int block_size){
    extern __shard__ float shared[];
    int idx = blockIdx.x; // range [0, B*T)
    int tid = threadIdx.x; // range [0, block_size)
    const float* x = inp + idx * C;
    // thread coarsening
    float sum = 0.0f;
    for (int  i = tid; i < C; i += block_size) {
        sum += x[i];
    }
    shared[tid] = sum;
    __syncthreads();
    // reductions
    for (int stride = block_size / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (tid < stride) {
            shared[tid] += shared[tid + stride];
        }
    }
    // write the final result (at thread 0) to global memory
    if (tid == 0) {
        mean[idx] = shared[0] / C;
    }
}

__global__ void normlization_kernel(float* out, const float* inp, float* mean, float* rstd, const float* weight, const float* bias, int B, int T, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int bt = idx / C;
    int c = idx % C;

    float m = mean[bt];
    float s = rstd[bt];
    float xi = inp[idx];
    float n = s * (xi - m);
    float o = n * weight[c] + bias[c];

    out[idx] = o;
}

__global__ void layernorm_forward3(float* __restrict__ out, float* __restrict__ rstd, const float* __restrict__ inp, const float* __restrict__ weight,
                                const float* __restrict__ bias, int N, int C) {
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    // meta_group_size is the number of warps in a block, and meta_group_rank is the warp index  
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    if (idx >= N) {
        return;
    }

    // the row of input that this group of threads is responsible for
    const float* x = inp + idx * C;

    // mean
    float sum = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()){
        sum += x[i];
    }
    sum = cg::reduce(warp, sum, cg::plus<float>{});
    float m = sum / C;
    if(warp.thread_rank() == 0 && mean != nullptr) {
        __stcs(mean + idx, m);
    }

    //rstd
    sum = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        float diff = x[i] - m;
        sum += diff * diff;
    }
    sum = cg::reduce(warp, sum, cg::plus<float>{});
    float s = rsqrtf(sum / C + 1e-5f);
    if(warp.thread_rank() == 0 && rstd != nullptr) {
        __stcs(rstd + idx, s);
    }

    float* o = out + idx * C;
    for (int c = warp.thread_rank(); c < C; c += warp.size()) {
        float n = s * (__ldcs(x+c) - m);
        __stcs(o+c, n * weight[c] + bias[c]);
    }
}

//block 级别的 reduce
auto block = cg::this_thread_block(); // definitions

auto warp32 = cg::tiled_partition<32>(block); // 32 thread warps
auto warp16 = cg::tiled_partition<16>(block); // 16 thread tiles
auto warp8 = cg::tiled_partition<8>(block); // 8 thread tiles
auto tile8 = cg::tiled_partition<8>(warp32); // 8 thread sub-warps
auto tile4 = cg::tiled_partition<4>(tile8); // 4 thread sub-sub warps

void layernorm_backward_cpu(float* dinp, float* dweight, float* dbias, const float* dout, const float* inp, const float* weight, int B, int T, int C) {
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            const float* dout_bt = dout + b * T * C + t * C;
            const float* inp_bt = inp + b * T * C + t * C;
            float* dinp_bt = dinp + b * T * C + t * C;
            const float mean_bt = mean[b * T + t];
            const float rstd_bt = rstd[b * T + t];

            // first: two reduce operations
            float dnorm_mean = 0.0f;
            float dnorm_norm_mean = 0.0f;
            for (int i = 0; i < C; i++) {
                float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
                float dnorm_i = weight[i] * dout_bt[i];
                dnorm_mean += dnorm_i;
                dnorm_norm_mean += dnorm_i * norm_bti;
            }
            dnorm_mean = dnorm_mean / C;
            dnorm_norm_mean = dnorm_norm_mean / C;

            // now iterate again and accumulate all the gradients
            for (int i = 0; i < C; i++) {
                float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
                float dnorm_i = weight[i] * dout_bt[i];
                // gradient contribution to bias
                dbias[i] += dout_bt[i];
                // gradient contribution to weight
                dweight[i] += norm_bti * dout_bt[i];
                // gradient contribution to input
                float dval = 0.0f;
                dval += dnorm_i; // term 1
                dval -= dnorm_mean; // term 2
                dval -= norm_bti * dnorm_norm_mean; //term 3
                dval *= rstd_bt; // final scale
                dinp_bt[i] += dval;
            }
        }
    }
}

int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx >= B*T) return;
int b = idx / T;
int t = idx % T;

const float* dout_bt = dout + b * T * C + t * C;
const float* inp_bt = inp + b * T * C + t * C;
float* dinp_bt = dinp + b * T * C + t * C;
const float mean_bt = mean[b * T + t];
const float rstd_bt = rstd[b * T + t];

// first: two reduce operaions
float dnorm_mean = 0.0f;
float dnorm_norm_mean = 0.0f;
for (int i = 0; i < C; i++) {
    float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
    float dnorm_i = weight[i] * dout_bt[i];
    dnorm_mean += dnorm_i;
    dnorm_norm_mean += dnorm_i * norm_bti;
}
dnorm_mean = dnorm_mean / C;
dnorm_norm_mean = dnorm_norm_mean / C;

// now iterate again and accumulate all the gradients
for (int i = 0; i < C; i++) {
    float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
    float dnorm_i = weight[i] * dout_bt[i];
    // gradient contribution to bias
    atomicAdd(&dbias[i], dout_bt[i]);
    // gradient contribution to weight
    atomicAdd(&dweight[i], norm_bti * dout_bt[i]);
    // gradient contribution to input
    float dval = 0.0f;
    dval += dnorm_i; // term 1
    dval -= dnorm_mean; // term 2
    dval -= norm_bti * dnorm_norm_mean; // term 3
    dval *= rstd_bt; // final scale
    dinp_bt[i] += dval;
}

extern __shared__ float shared[]; // size = 2 * C

namespace cg = cooperative_groups;
cg::thread_block block = cg::this_thread_block();
cg::thread_block_tile<32> warp = cg::this_thread_block();
int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
int N = B * T;
if(idx >= N) { return; } // thread guards

int b = idx / T;
int t = idx % T;

const float* dout_bt = dout + b * T * C + t * C;
const float* inp_bt = inp + b * T * C + t * C;
float* dinp_bt = dinp + b * T * C + t * C;
const float mean_bt = mean[b * T + t];
const float rstd_bt = rstd[b * T + t];

// the first half of shared memory is bias, second is weight
float* dbias_shared = shared;
float* dweight_shared = shared + C;

// init shared memory to zero
#pragma unroll
    for(int i = threadIdx.x; i < C; i+= blockDim.x){
        dbias_shared[i] = 0.0f;
        dweight_shared[i] = 0.0f;
    }
    __syncthreads();

    // first: two reduce operations
    float dnorm_mean = 0.0f;
    float dnorm_norm_mean = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
        float dnorm_i = weight[i] * dout_bt[i];
        dnorm_mean += dnorm_i;
        dnorm_norm_mean += dnorm_i * norm_bti;
    }
    dnorm_mean = cg::reduce(warp, dnorm_mean, cg::plus<float>{});
    dnorm_norm_mean = cg::reduce(warp, dnorm_norm_mean, cg::plus<float>{});
    dnorm_mean = dnorm_mean / C;
    dnorm_norm_mean = dnorm_norm_mean / C;

    // now iterate again and accumulate all the gradients
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
        float dnorm_i = weight[i] * dout_bt[i];
        // gradient contribution to bias
        atomicAdd(&dbias_shared[i], dout_bt[i]);
        // gradient contribution to weight
        atomicAdd(&dweight_shared[i], norm_bti * dout_bt[i]);
        // gradient contribution to input
        float dval = 0.0f;
        dval += dnorm_i; // term 1
        dval -= dnorm_mean; // term 2
        dval -= norm_bti * dnorm_norm_mean; // term 3
        dval *= rstd_bt; // final scale
        dinp_bt[i] += dval;
    }
    __syncthreads();

    // write to global memory
        for(int i = threadIdx.x; i < C; i += blockDim.x){
            atomicAdd(&dbias[i], dbias_shared[i]);
            atomicAdd(&dweight[i], dweight_shared[i]);
        }