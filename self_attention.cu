#include "hip/hip_runtime.h"
void attention_forward_cpu(float* out, float* preatt, float* att, const float* inp, int B, int T, int C, int NH) {
    // input is (B, T, 3C) Q,K,V
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int C3 = C*3;
    int hs = C / NH; // head size
    float scale = 1.0 / sqrtf(hs);

    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            for (int h = 0; h < NH; h++) {
                const float* query_t = inp + b * T * C3 + t * C3 + h * hs;
                float* preatt_bth = preatt + b*NH*T*T + h*T*T + t*T;
                float* att_bth = att + b*NH*T*T + h*T*T + t*T;

                // pass 1: calculate query dot key and maxval
                float maxval = -10000.0f;
                for (int t2 = 0; t2 <= t; t2++) {
                    const float* key_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C; // +C because it's key

                    // (query_t) dot (key_t2)
                    float val = 0.0f;
                    for (int i = 0; i < hs; i++) {
                        val += query_t[i] * key_t2[i];
                    }
                    val *= scale;
                    if (val > maxval) {
                        maxval = val;
                    }

                    preatt_bth[t2] = val;
                }
                // pad with -INFINITY outside of autoregressive region for debugging comparisons
                for (int t2 = t+1; t2 < T; t2++){
                    preatt_bth[t2] = -INFINITY;
                }

                // pass 2: calculate the exp and keep track of sum
                float expsum = 0.0f;
                for (int t2 = 0; t2 <= t; t2++) {
                    float expv = expf(preatt_bth[t2] - maxval);
                    expsum += expv;
                    att_bth[t2] = expv;
                }
                float expsum_inv = expsum == 0.0f ? 0.0F : 1.0f / expsum;

                // pass 3: normalize to get the softmax
                for (int t2 = 0; t2 < T; t2++) {
                    if (t2 <= t) {
                        att_bth[t2] *= expsum_inv;
                    } else {
                        // causal attention mask. not strictly necessary to set to zero here
                        // only doing this explicitly for debugging and checking to Pytorch
                        att_bth[t2] = 0.0f;
                    }
                }
    // pass 4: accumulate weighted values into the output of attention
    float* out_bth = out + b * T * C + t * C + h * hs;
    for (int i = 0; i < hs; i++) { out_bth[i] = 0.0f; }
    for (int t2 = 0; t2 <=t; t2++) {
        const float* value_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C * 2;
        float att_btht2 = att_bth[t2];
        for (int i = 0; i < hs; i++) {
            out_bth[i] += att_btht2 * value_t2[i];
        }
    }
            }
        }
    }
}

// flash attention simple version

// these are hardcoded to 32 for now
const int Bc = 32;
const int Br = 32;
// renaming these to be consistent with the kernel
// const int B = B;
const int nh = NH;
const int N = T;
const int d = C / NH;
// more
const int Tc = ceil((float) N / Bc);
const int Tr = ceil((float) N / Br);
const float softmax_scale = 1.0 / sqrt(d);

// calculate SRAM size needed per block, ensure we have enough shared memory
int col_tile_size = Bc * d; // size of Kj, Vj
int row_tile_size = Br * d; // size of Qi
const int sram_size = 
    (2 * col_tile_size * sizeof(float)) // SRAM size for Kj, Vj
    + (row_tile_size * sizeof(float)) // SRAM size for Qi
    + (Bc * Br * sizeof(float)); // SRAM size for S
int max_sram_size;
hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
if (sram_size > max_sram_size) {
    printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);
    printf("SRAM size exceeds maximum shared memory per block\n");
    printf("Try decreasing col_tile_size or row_tile_size further\n");
    exit(1);
}

//permute kernel
__global__ void permute_kernel(float* q, float* k, float* v, const float* inp, int B, int N, int NH, int d) {
    // so now, this kernel wants Q,K,V to all be of shape (B, NH, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, NH, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Q[b][nh_][n][d_] = inp[b][n][0][nh_][d_]

    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int inp_idx = \
            (b * N * 3 * NH * d)
            +   (N * 3 * NH * d)
            +       (0 * NH * d)
            +          (nh_ * d)
            +                d_;

        q[idx] = inp[inp_idx];
        k[idx] = inp[inp_idx + NH * d];
        v[idx] = inp[inp_idx + 2 * (NH * d)];
    }
}

__global__ void attention_forward_kernel2(
    const float* Q,
    const float* K,
    const float* V,
    const int N,
    const int d,
    const int Tc,
    const int Tr,
    const int Bc,
    const int Br,
    const float softmax_scale,
    float* l,
    float* m,
    float* O
) {
    int tx = threadIdx.x;
    int bx = blockIdx.x; int by = blockIdx.y; //batch and head index

    // Offset into Q,K,V,O,L,m - different for each batch and head
    int qkv_offset = (bx * gridDim.y * N * d) = (by * N * d); // gridDim.y = nh
    int lm_offset = (bx * gridDim.y * N) + (by * N); // offset for l and m

    // Define SRAM for Q,K,V,S
    extern __shared__ float sram[];
    int tile_size = Bc * d; // size of Qi, Kj, Vj
    float* Qi = sram;
    float* Kj = &sram[tile_size];
    float* Vj = &sram[tile_size * 2];
    float* S = &sram[tile_size * 3];

    for (int j = 0; j < Tc; j++) {

        // Load Kj, Vj to SRAM
        for (int x = 0; x < d; x++) {
            Kj[(tx * d) + x] = K[qkv_offset + (tile_size * j) + (tx * d) + x];
            Vj[(tx * d) + x] = V[qkv_offset + (tile_size * j) + (tx * d) + x];
        }
        __syncthreads(); // such that the inner loop can use the correct Kj, Vj

        for (int i = 0; i < Tr; i++) {
            // if past the end of the sequence, break
            if (i * Br + tx >= N) {
                break;
            }

            // Load Qi to SRAM, L and m to registers
            for (int x = 0; x < d; x++) {
                Qi[(tx * d) + x] = Q[qkv_offset + (tile_size * i) + (tx * d) +  x]
            }
            float row_m_prev = m[lm_offset + (Br * i) + tx];
            float row_l_prev = l[lm_offset + (Br * i) + tx];

            // S = QK^T, row_m = rowmax(S)
            // S[tx][y] = Sum_{x = 0}^{d-1} {Qi[tx][x] * kj[y][x]}
            // row_m = Max_{y = 0}&{Bc-1} S[tx][y]
            // with causal masking
            float row_m = -INFINITY;
            for (int y = 0; y < Bc; y++) {
                if (j * Bc + y >= N) {
                    break;
                }
                float sum = 0;
                for (int x = 0; x < d; x++) {
                    sum += Qi[(tx * d) + x] * Kj[(y * d) + x];
                }
                sum *= softmax_scale;
                if (i * Br + tx < j * Bc + y)
                    sum = -INFINITY;
                S[(Bc * tx) + y] = sum;

                if (sum > row_m)
                    row_m = sum;
            }

            // implement softmax with causal masking
            // P = exp(S - row_m), row_l = rowsum(P)
            // P[tx][y] = exp(S[tx][y] - row_m)
            float row_l = 0;
            for (int y = 0; y < Bc; y++) {
                if (j * Bc + y >= N) {
                    break;
                }
                if (i * Br + tx < j * Bc + y)
                    S[(Bc * tx) + y] = 0;
                else
                    S[(Bc * tx) + y] = __expf(S[(Bc * tx) + y] - row_m);
                row_l += S[(Bc * tx) + y];
            }

            // Compute new m and l
            float row_m_new = max(row_m_prev, row_m);
            float row_l_new = (__expf(row_m_prev - row_m_new) * row_l_prev) + (__expf(row_m - row_m_new) * row_l);

            // Write O, l, m to HBM
            for (int x = 0; x < d; x++) {
                float pv = 0;
                for (int y = 0; y < Bc; y++) {
                    if (j * Bc + y >= N) {
                        break;
                    }
                    pv += S[(Bc * tx) + y] * Vj[(y * d) + x];
                }
                O[qkv_offset + (tile_size * i) + (tx * d) + x] = (1 / row_l_new)
                    * ((row_l_prev * __expf(row_m_prev - row_m_new) * O[qkv_offset + (tile_size * i) + (tx * d) + x]) \
                    + (__expf(row_m - row_m_new) * pv));
            }
            m[lm_offset + (Br * i) + tx] = row_m_new;
            l[lm_offset + (Br * i) + tx] = row_l_new;
        }
        __syncthreads(); // otherwise, thread can use the wrong Kj, Vj in inner loop
    }
}

__global__ void unpermute_kernel(const float* inp, float * out, int B, int N, int NH, int d) {
    // out hhas shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)
    int idx = blockIdx.x * blockDIm.x + threadIdx.x;

    // out[b][n][nh_][d_] <- inp[b][nh_][n][d_]
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
        out[other_idx] = inp[idx];
    }
}

// cublas
// batched matrix multiply with cuBLAS
const float alpha = 1.0f;
const foat beta = 0.0f;
cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, T, T, HS, &alpha, k,
                                        Hs, T * HS, q, HS, T * HS, &beta, preatt, T, T * T, B * NH));
// new approach:first cuBLAS another batched matmul
// y = att @ v # (B, nh, T, T) @ (B, nh, T, hs) -> (B, nh, T, hs)
cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                        HS, T, T,
                                        &alpha,
                                        v, HS, T * HS,
                                        att, T, T * T,
                                        &beta,
                                        vaccum, HS, T * HS,
                                        B * NH));